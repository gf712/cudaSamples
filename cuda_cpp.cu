//
// Created by gil on 29/05/18.
//

#include "cuda_cpp.h"
#include "cuda_kernels.h"
#include <iostream>

void square_gpu(float *h_in, float *h_out, int ARRAY_SIZE, int ARRAY_BYTES) {

    // C pointers to point to CUDA memory
    float *d_in;
    float *d_out;

    // allocate GPU memory
    // hipMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // copy array from CPU to GPU
    // hipMemcpy(GPU pointer, CPU pointer, pointer size, hipMemcpyHostToDevice/hipMemcpyDeviceToHost/hipMemcpyDeviceToDevice)
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch kernel with GPU pointer
    //   -> ARRAY_SIZE+THREADS_PER_BLOCK-1 ensures that we always launch at least one thread block for N > 0
    square <<< (ARRAY_SIZE+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (d_in, d_out);

    // copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);
}


void vector_dot_product_gpu(float *h_u, float* h_v, float *h_out, int ARRAY_SIZE, int ARRAY_BYTES) {

    // C pointers to point to CUDA memory
    float *d_v;
    float *d_u;
    float *d_out;

    // allocate GPU memory
    // hipMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void **) &d_v, ARRAY_BYTES);
    hipMalloc((void **) &d_u, ARRAY_BYTES);
    // store result (a scalar) in a one element array -> size = sizeof(float)
    hipMalloc((void **) &d_out, sizeof(float));

    // copy array from CPU to GPU
    // hipMemcpy(GPU pointer, CPU pointer, pointer size, hipMemcpyHostToDevice/hipMemcpyDeviceToHost/hipMemcpyDeviceToDevice)
    hipMemcpy(d_v, h_v, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_u, h_u, ARRAY_BYTES, hipMemcpyHostToDevice);
//    hipMemcpy(d_out, h_out, sizeof(float), hipMemcpyHostToDevice);

    // launch kernel with GPU pointer
    //   -> ARRAY_SIZE+THREADS_PER_BLOCK-1 ensures that we always launch at least one thread block for N > 0
    vector_dot_product <<< (ARRAY_SIZE+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (d_u, d_v, d_out);

    // copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(d_v);
    hipFree(d_u);
    hipFree(d_out);
}