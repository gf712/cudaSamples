//
// Created by gil on 29/05/18.
//

#include "cuda_cpp.h"
#include "cuda_kernels.h"

void square_cpp(float *h_in, float *h_out, int ARRAY_SIZE, int ARRAY_BYTES) {

// C pointers to point to CUDA memory
    float *d_in;
    float *d_out;

// allocate GPU memory
// hipMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

// copy array from CPU to GPU
// hipMemcpy(GPU pointer, CPU pointer, pointer size, hipMemcpyHostToDevice/hipMemcpyDeviceToHost/hipMemcpyDeviceToDevice)
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

// launch kernel with GPU pointer
    square <<< 1, ARRAY_SIZE >>> (d_in, d_out);

// copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

// free cuda memory
    hipFree(d_in);
    hipFree(d_out);
}