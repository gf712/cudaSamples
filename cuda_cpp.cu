//
// Created by gil on 29/05/18.
//

#include "cuda_cpp.h"
#include "cuda_kernels.h"
#include <iostream>

void square_gpu(float *h_in, float *h_out, int ARRAY_SIZE, int ARRAY_BYTES) {

    // C pointers to point to CUDA memory
    float *d_in;
    float *d_out;

    // allocate GPU memory
    // hipMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // copy array from CPU to GPU
    // hipMemcpy(GPU pointer, CPU pointer, pointer size, hipMemcpyHostToDevice/hipMemcpyDeviceToHost/hipMemcpyDeviceToDevice)
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    int n_threads = ARRAY_SIZE / THREADS_PER_BLOCK > 0? ARRAY_SIZE / THREADS_PER_BLOCK : 1;

    // launch kernel with GPU pointer
    square <<< n_threads, THREADS_PER_BLOCK>>> (d_in, d_out);

    // copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);
}