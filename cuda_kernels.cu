#include "hip/hip_runtime.h"
//
// Created by gil on 29/05/18.
//
#include "cuda_kernels.h"

__global__ void square(float* d_in, float* d_out) {

    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;

}

__global__ void vector_dot_product(float* u, float* v, float* out) {



}
