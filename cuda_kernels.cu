#include "hip/hip_runtime.h"
//
// Created by gil on 29/05/18.
//
#include "cuda_kernels.h"

__global__ void square(float* d_in, float* d_out) {

    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;

}
