#include "hip/hip_runtime.h"
//
// Created by gil on 29/05/18.
//
#include <>
#include "cuda_kernels.h"
#include "cuda_cpp.h"

__global__ void square(float* d_in, float* d_out) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

__global__ void vector_dot_product(float* d_u, float* d_v, float* d_out) {

    // each block of threads has its own shared memory,
    // so can declare an array where we store results accessible by all the threads in block
    __shared__ float cache[THREADS_PER_BLOCK];

    // first calculate element product
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    cache[threadIdx.x] = d_u[idx] * d_v[idx];

    __syncthreads();

    // thread 0 will be responsible for adding the results
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; ++i) {
            sum += cache[i];
        }
        atomicAdd(d_out, sum);
    }
}
