//
// Created by gil on 18/12/17.
//
#include <cstdio>
#include "cuda_kernels.h"

int main() {

    int ARRAY_SIZE = 64;
    int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // initialise array (in CPU)
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        h_in[i] = static_cast<float>(i);
    }
    float h_out[ARRAY_SIZE];

    // C pointers to point to CUDA memory
    float* d_in;
    float* d_out;

    // allocate GPU memory
    // hipMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    // copy array from CPU to GPU
    // hipMemcpy(GPU pointer, CPU pointer, pointer size, hipMemcpyHostToDevice/hipMemcpyDeviceToHost/hipMemcpyDeviceToDevice)
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch kernel with GPU pointer
    square <<<1, ARRAY_SIZE>>>(d_in, d_out);

    // copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print result
    for (int j = 0; j < ARRAY_SIZE; ++j) {
        if (j == 0) {
            printf("[%f\t", h_out[j]);
        }

        else {
            printf(" %f", h_out[j]);

            if (j == ARRAY_SIZE - 1) {
                printf("]\n");
            } else if ((j + 1) % 4 == 0 && j > 0) {
                printf(",\n");
            } else {
                printf(",\t");
            }
        }
    }

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}