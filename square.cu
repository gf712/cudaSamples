//
// Created by gil on 18/12/17.
//


#include <hip/hip_runtime.h>
#include <cstdio>

// helps out with highlighting in CLion IDE
// https://stackoverflow.com/questions/39980645/enable-code-indexing-of-cuda-in-clion
#ifdef __JETBRAINS_IDE__
    #define __host__
    #define __device__
    #define __shared__
    #define __constant__
    #define __global__
    #define cudaMemcpyHostToDevice
    #define cudaMemcpyDeviceToHost

    typedef unsigned int uint;

    typedef struct uint3{
        uint x;
        uint y;
        uint z;
    }uint3;

    extern uint3 threadIdx;
#endif

__global__ void square(float* d_in, float* d_out) {

    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;

}

int main() {

    int ARRAY_SIZE = 64;
    int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // initialise array (in CPU)
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        h_in[i] = static_cast<float>(i);
    }
    float h_out[ARRAY_SIZE];

    // C pointers to point to CUDA memory
    float* d_in;
    float* d_out;

    // allocate GPU memory
    // cudaMalloc((void**) C pointer, size of C pointer)
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    // copy array from CPU to GPU
    // cudaMemcpy(GPU pointer, CPU pointer, pointer size, cudaMemcpyHostToDevice/cudaMemcpyDeviceToHost/cudaMemcpyDeviceToDevice)
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch kernel with GPU pointer
    square <<<1, ARRAY_SIZE>>>(d_in, d_out);

    // copy GPU pointer to CPU pointer
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print result
    for (int j = 0; j < ARRAY_SIZE; ++j) {
        if (j == 0) {
            printf("[%f\t", h_out[j]);
        }

        else {
            printf(" %f", h_out[j]);

            if (j == ARRAY_SIZE - 1) {
                printf("]\n");
            } else if ((j + 1) % 4 == 0 && j > 0) {
                printf(",\n");
            } else {
                printf(",\t");
            }
        }
    }

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}